
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_form_gpu()
{
    printf("hello World form the GPU\n");
}

int main(void) 
{
    hello_form_gpu<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
